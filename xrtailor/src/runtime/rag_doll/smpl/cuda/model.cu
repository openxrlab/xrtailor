#include <iostream>

#include <xrtailor/runtime/rag_doll/smpl/smplx.hpp>
#include <xrtailor/runtime/rag_doll/smpl/internal/cuda_util.cuh>

namespace smplx {

namespace {
using cuda_util::from_host_eigen_matrix;
using cuda_util::from_host_eigen_sparse_matrix;
}  // namespace

template <class ModelConfig>
__host__ void Model<ModelConfig>::_cuda_load() {
  from_host_eigen_matrix(device.verts, verts);
  from_host_eigen_matrix(device.blend_shapes, blend_shapes);

  from_host_eigen_sparse_matrix(device.joint_reg, joint_reg);
  {
    SparseMatrix tmp_weights = weights;  // Change to CSR
    from_host_eigen_sparse_matrix(device.weights, tmp_weights);
  }

  if (n_hand_pca) {
    from_host_eigen_matrix(device.hand_comps_l, hand_comps_l);
    from_host_eigen_matrix(device.hand_comps_r, hand_comps_r);
    from_host_eigen_matrix(device.hand_mean_l, hand_mean_l);
    from_host_eigen_matrix(device.hand_mean_r, hand_mean_r);
  }
}
template <class ModelConfig>
__host__ void Model<ModelConfig>::_cuda_copy_template() {
  const size_t dsize = verts.size() * sizeof(verts.data()[0]);
  hipMemcpy(device.verts, verts.data(), dsize, hipMemcpyHostToDevice);
}
template <class ModelConfig>
__host__ void Model<ModelConfig>::_cuda_free() {
  if (device.verts)
    hipFree(device.verts);
  if (device.blend_shapes)
    hipFree(device.blend_shapes);
  if (device.joint_reg_dense)
    hipFree(device.joint_reg_dense);
  if (device.joint_reg.values)
    hipFree(device.joint_reg.values);
  if (device.joint_reg.inner)
    hipFree(device.joint_reg.inner);
  if (device.joint_reg.outer)
    hipFree(device.joint_reg.outer);
  if (device.weights.values)
    hipFree(device.weights.values);
  if (device.weights.inner)
    hipFree(device.weights.inner);
  if (device.weights.outer)
    hipFree(device.weights.outer);
  if (device.hand_comps_l)
    hipFree(device.hand_comps_l);
  if (device.hand_comps_r)
    hipFree(device.hand_comps_r);
  if (device.hand_mean_l)
    hipFree(device.hand_mean_l);
  if (device.hand_mean_r)
    hipFree(device.hand_mean_r);
}

// Instantiation
template class Model<model_config::SMPL>;
template class Body<model_config::SMPL_v1>;
template class Model<model_config::SMPLH>;
template class Model<model_config::SMPLX>;
template class Model<model_config::SMPLXpca>;
template class Model<model_config::SMPLX_v1>;
template class Model<model_config::SMPLXpca_v1>;

}  // namespace smplx
